#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "helper.h"

#define GPU_RUNS 100
 
__global__ void mul2Kernel(float* X, float *Y, int N) {
    const unsigned int gid_x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int gid_y = gid_x+N;
    if (gid_x < N) Y[gid_x] = X[gid_x] * X[gid_y];
}

int main(int argc, char** argv) {
    unsigned int N;
    
    { // reading the number of elements 
      if (argc != 2) { 
        printf("Num Args is: %d instead of 1. Exiting!\n", argc); 
        exit(1);
      }

      N = atoi(argv[1]);
      printf("N is: %d\n", N);

      const unsigned int maxN = 500000000;
      if(N > maxN) {
          printf("N is too big; maximal value is %d. Exiting!\n", maxN);
          exit(2);
      }
    }

    // use the first CUDA device:
    hipSetDevice(0);

    unsigned int mem_size = N*sizeof(float);

    // allocate host memory for both CPU and GPU
    float* h_in  = (float*) malloc(mem_size*2);
    float* gpu_res = (float*) malloc(mem_size);
    float* cpu_res = (float*) malloc(mem_size);

    // initialize the memory
    for(unsigned int i=0; i<N; ++i) {
        h_in[i] = (float)i;
        h_in[i+N] = (float)i*2;
    }


    // sequential map on CPU
    for (unsigned int i = 0; i < N; i++){
        cpu_res[i] = h_in[i] * h_in[i+N];
    }


    // allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    unsigned int B = 256;
    unsigned int numblocks = (N + B - 1) / B;
    
    // a small number of dry runs
    for(int r = 0; r < 1; r++) {
        dim3 block(B, 1, 1), grid(numblocks, 1, 1);
        mul2Kernel<<< grid, block>>>(d_in, d_out, N);
    }
  
    { // execute the kernel a number of times;
      // to measure performance use a large N, e.g., 200000000,
      // and increase GPU_RUNS to 100 or more. 
    
        double elapsed; struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        for(int r = 0; r < GPU_RUNS; r++) {
            dim3 block(B, 1, 1), grid(numblocks, 1, 1);
            mul2Kernel<<< grid, block>>>(d_in, d_out, N);
        }
        hipDeviceSynchronize();
        // ^ `hipDeviceSynchronize` is needed for runtime
        //     measurements, since CUDA kernels are executed
        //     asynchronously, i.e., the CPU does not wait
        //     for the kernel to finish.
        //   However, `hipDeviceSynchronize` is expensive
        //     so we need to amortize it across many runs;
        //     hence, when measuring performance use a big
        //     N and increase GPU_RUNS to 100 or more.
        //   Sure, it would be better by using CUDA events, but
        //     the current procedure is simple & works well enough.
        //   Please note that the execution of multiple
        //     kernels in Cuda executes correctly without such
        //     explicit synchronization; we need this only for
        //     runtime measurement.
        
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (1.0 * (t_diff.tv_sec*1e6+t_diff.tv_usec)) / GPU_RUNS;
        double gigabytespersec = (2.0 * N * 4.0) / (elapsed * 1000.0);
        printf("The kernel took on average %f microseconds. GB/sec: %f \n", elapsed, gigabytespersec);
        
    }
        
    // check for errors
    gpuAssert( hipPeekAtLastError() );

    // copy result from device to host

    hipMemcpy(gpu_res, d_out, mem_size, hipMemcpyDeviceToHost);

    // element-wise compare of CPU and GPU execution
    for (unsigned int i = 0; i < N; i++) {
        float actual = cpu_res[i];
        float expected = gpu_res[i];
        if (fabs(actual - expected >= 0.000001)) {
            printf("Invalid result at index %d, actual: %f, expected: %f. \n", i, actual, expected);
            exit(3);
        }
    }

    printf("Successful Validation.\n");

    // clean-up memory
    free(h_in);       free(gpu_res);
    hipFree(d_in);   hipFree(d_out);
}
